// #include "../include/matrix.cu"
// #include "../include/matmul.cu"

#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>
#include <cstdlib>
#include <cstdint>	
#include <filesystem>
#include <fstream>
#include <iostream>
#include <string>
#include <thread>

// row-major order
#define IDX2D(i, j, N) (((i)*(N))+(j))

// Device code
__global__ void InitializeMatrices(float *C, float *A, float *B, const int N, const unsigned long long seed){
	int tidx = threadIdx.x + blockIdx.x * blockDim.x;
	int tidy = threadIdx.y + blockIdx.y * blockDim.y; 
	int xthreads = gridDim.x * blockDim.x;
	int ythreads = gridDim.y * blockDim.y; 

	// Generate random number b/w [0.0, 1.0]
    hiprandState_t state;
    if (tidx < N && tidy < N){
        hiprand_init(seed, tidx, 0, &state);
    }

	for (int i = tidx; i < N; i += xthreads){
		for (int j = tidy; j < N; j += ythreads){
			A[IDX2D(i, j, N)] = static_cast<float>(hiprand_uniform(&state));
			B[IDX2D(i, j, N)] = static_cast<float>(hiprand_uniform(&state));
			C[IDX2D(i, j, N)] = 0.0;
		}
	}
	return;
}

__global__ void MatMul(float *C, const float *A, const float *B, const int N){ 
	int tidx = threadIdx.x + blockIdx.x * blockDim.x;
	int tidy = threadIdx.y + blockIdx.y * blockDim.y; 
	int xthreads = gridDim.x * blockDim.x;
	int ythreads = gridDim.y * blockDim.y; 

	float sum;
	for (int i = tidx; i < N; i += xthreads){
		for (int j = tidy; j < N; j += ythreads){
			sum = 0.0;
			for (int k = 0; k < N; k++){
				sum += A[IDX2D(i, k, N)] * B[IDX2D(k, j, N)];
			}
			C[IDX2D(i, j, N)] = sum;
		}
	}
	return;
} 

/* Host Code */
void hostMatMul(float* C, const float *A, const float *B, const int N, const int begin, const int end){
    // row-major storage
    float sum;
    for (int i = begin; i < end; i++){ 
        for (int j = begin; j < end; j++){
            sum = 0.0;
            for (int k = 0; k < N; k++){
                sum += A[IDX2D(i, j, N)] * B[IDX2D(i, j, N)];
            }
            C[IDX2D(i, j, N)] = sum;
        }
    }
    return;
}

// Validating on host because arrays get relatively large on GPU, and it's simpler to not have to weave freeing up space with getting work done 
void hostValidate(const float* d_C, const float *h_C, const int N, bool *are_same, const int begin, const int end){
    *are_same = true; // don't want compiler optimizing its way through the validation check
    float threshold = 0.000001;
    for (int i = begin; i < end; i++){
        for (int j = begin; j < end; j++){
            if (abs(d_C[IDX2D(i, j, N)] - h_C[IDX2D(i, j, N)]) > threshold){
                std::cout << "(i,j) = (" << i << "," << j << ")" << std::endl;
                std::cout << "d_C = " << d_C[IDX2D(i, j, N)] << std::endl;
                std::cout << "h_C = " << h_C[IDX2D(i, j, N)] << std::endl;
                *are_same = false;
                break;
            }
        }
    }
    return;
}

// https://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
#define checkCuda(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

int main(int argc, char* argv[]){
	// Accept arguments 
	// int N = atoi(argv[1]); // length of matrix side - CHECK IF THIS IS CAUSING BUG
	uint64_t N = atoll(argv[1]);
	int SM_multiplier_x = atoi(argv[2]); // used for changing number of blocks
	int SM_multiplier_y = atoi(argv[3]);
	int num_threads_per_block_x = atoi(argv[4]);
	int num_threads_per_block_y = atoi(argv[5]);

	// Allocate device matrices
	float *A, *B, *C; // flattened arrays because that is easiest with CUDA
    
	uint64_t requested_matrix_memory = N*N*sizeof(float);

	checkCuda(hipMalloc(&A, requested_matrix_memory));
	checkCuda(hipMalloc(&B, requested_matrix_memory));
	checkCuda(hipMalloc(&C, requested_matrix_memory));

    // Allocate host matrices
    float *h_A, *h_B, *h_C; 

    h_A = (float*)malloc(requested_matrix_memory);
    h_B = (float*)malloc(requested_matrix_memory);
    h_C = (float*)malloc(requested_matrix_memory);

	std::cout << "Size of matrices is: " << pow(N,2) << std::endl;

    // Get device attributes 
    int deviceId;
    int numberOfSMs;

    hipGetDevice(&deviceId);
    hipDeviceGetAttribute(&numberOfSMs, hipDeviceAttributeMultiprocessorCount, deviceId);

	// Define execution configuration
	dim3 block_dimensions(num_threads_per_block_x, num_threads_per_block_y);
	dim3 grid_dimensions(numberOfSMs * SM_multiplier_x, numberOfSMs * SM_multiplier_y);

	// Set up timer
	hipEvent_t start_search, stop_search;
    hipEventCreate(&start_search);
    hipEventCreate(&stop_search);
    float time_search;

	// Initialize Matrices
	InitializeMatrices<<<block_dimensions, grid_dimensions>>>(C, A, B, N, 1234); // Magic number at the end is seed for rng
	checkCuda(hipDeviceSynchronize());

    // Copy device data to host
    checkCuda(hipMemcpy(h_A, A, requested_matrix_memory, hipMemcpyDeviceToHost));
    checkCuda(hipMemcpy(h_B, B, requested_matrix_memory, hipMemcpyDeviceToHost));
    checkCuda(hipMemcpy(h_C, C, requested_matrix_memory, hipMemcpyDeviceToHost));
    checkCuda(hipDeviceSynchronize());

    // Perform Matrix Multiplication on Host
    std::thread t1(hostMatMul, h_C, h_A, h_B, N, 0, N/4);
    std::thread t2(hostMatMul, h_C, h_A, h_B, N, N/4, N/2);
	std::thread t3(hostMatMul, h_C, h_A, h_B, N, N/2, 3*N/4);
    std::thread t4(hostMatMul, h_C, h_A, h_B, N, 3*N/4, N);

    t1.join(); t2.join(); t3.join(); t4.join();

    // Perform Matrix Multiplication on Device
	hipEventRecord(start_search, 0);
	MatMul<<<block_dimensions, grid_dimensions>>>(C, A, B, N);
	hipEventRecord(stop_search, 0);
	hipEventSynchronize(stop_search);
	hipEventElapsedTime(&time_search, start_search, stop_search);

	std::cout << "Elapsed time is: " << time_search << " ms" << std::endl;

    // Validate that matrices are the same
    float *v_C; 

    v_C = (float*)malloc(requested_matrix_memory);
    checkCuda(hipMemcpy(v_C, C, requested_matrix_memory, hipMemcpyDeviceToHost));
    checkCuda(hipDeviceSynchronize());

    bool are_same1, are_same2, are_same3, are_same4; // simpler than using lambdas and <future>

    std::thread v1(hostValidate, v_C, h_C, N, &are_same1, 0, N/4);
    std::thread v2(hostValidate, v_C, h_C, N, &are_same2, N/4, N/2);
    std::thread v3(hostValidate, v_C, h_C, N, &are_same3, N/2, 3*N/4);
    std::thread v4(hostValidate, v_C, h_C, N, &are_same4, 3*N/4, N);

    v1.join(); v2.join(); v3.join(); v4.join();

    if (are_same1 && are_same2 && are_same3 && are_same4){
        std::cout << "h_C and d_C are the same :) " << std::endl;
    }
    else {
        std::cout << "h_C and d_C not the same :( " << std::endl;
        std::cout << "are_same 1 = " << are_same1 << std::endl;
        std::cout << "are_same 2 = " << are_same2 << std::endl;
        std::cout << "are_same 3 = " << are_same3 << std::endl;
        std::cout << "are_same 4 = " << are_same4 << std::endl;
    }

	// Free data
	hipFree(A);
	hipFree(B);
	hipFree(C);
    free(h_A);
    free(h_B);
    free(h_C);
    free(v_C);
	return 0;
}